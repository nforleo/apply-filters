#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 128

#include <cmath>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include "tiffio.h"

// saves TIFF file from data in `raster`
void save_tiff(const char *fname, uint32 *raster, uint32 w, uint32 h) {
    TIFF *tif = TIFFOpen(fname, "w");
    if (! raster) {
        throw std::runtime_error("Could not open output file");
    }
    TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, w);
    TIFFSetField(tif, TIFFTAG_IMAGELENGTH, h);
    TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 4);
    TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, 8);
    TIFFSetField(tif, TIFFTAG_COMPRESSION, COMPRESSION_DEFLATE);
    TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);
    TIFFSetField(tif, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_RGB);
    TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
    TIFFWriteEncodedStrip(tif, 0, raster, w*h*4);
    TIFFClose(tif);
}

// loads image data from `fname` (allocating dynamic memory)
// *w and *h are updated with the image dimensions
// raster is a matrix flattened into an array using row-major order
// every uint32 in the array is 4 bytes, enconding 8-bit packed ABGR
// A: transparency attribute (can be ignored)
// B: blue pixel
// G: green pixel
// R: red pixel
uint32 *load_tiff(const char *fname, uint32 *w, uint32 *h) {
    TIFF *tif = TIFFOpen(fname, "r");
    if (! tif) {
        throw std::runtime_error("Could not open input file");
    }
    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, w);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, h);
    uint32 *raster = (uint32 *) _TIFFmalloc(*w * *h * sizeof (uint32));
    if (! raster) {
        TIFFClose(tif);
        throw std::runtime_error("Memory allocation error");
    }
    if (! TIFFReadRGBAImageOriented(tif, *w, *h, raster, ORIENTATION_TOPLEFT, 0)) {
        TIFFClose(tif);
        throw std::runtime_error("Could not read raster from TIFF image");
    }
	
    TIFFClose(tif);
    return raster;
}

void clamp(float *val) {
    if (*val < 0) *val = 0;
    if (*val > 255) *val = 255;
}

__device__ void cuda_clamp(float *val) {
    if (*val < 0) *val = 0;
    if (*val > 255) *val = 255;
}

void filter_image_seq(uint32 *raster, uint32 w, uint32 h, const float *filter, int f_len) {
    // to get RGB values from a pixel, you can either use bitwise masks
    // or rely on the following macros:
    // TIFFGetR(raster[i]) red
    // TIFFGetG(raster[i]) green
    // TIFFGetB(raster[i]) blue 
    // TIFFGetA(raster[i]) this value should be ignored
    //
    // to modify RGB values from a pixel, you can use bitwise shifts or masks
    // each pixel stores values in the order ABGR
    //
    // TODO: here you will filter the image in raster
    //
    uint32 *copy = new uint32[w*h];
    std::memcpy(copy, raster, sizeof(uint32)*w*h);
    uint32 d = (uint32) std::sqrt(f_len);
    uint32 idx, pixel;
    uint32 st = d / 2;
    uint32 end_w = w - d/2;
    uint32 end_h = h - d/2;
    float sumR, sumG, sumB;
    // applies filter
    for (uint32 i = st ; i < end_h ; i++) {
        for (uint32 j = st ; j < end_w ; j++) {
            sumR = sumG = sumB = 0;
            for (uint32 k = 0 ; k < d ; k ++) {
                idx = (i-st+k)*w + (j-st);
                for (uint32 l = 0 ; l < d ; l++) {
                    pixel = copy[idx++];
                    sumR += (filter[k*d + l] * TIFFGetR(pixel));
                    sumG += (filter[k*d + l] * TIFFGetG(pixel));
                    sumB += (filter[k*d + l] * TIFFGetB(pixel));
                }
            }
            clamp(&sumR);
            clamp(&sumG);
            clamp(&sumB);
            raster[i*w + j] = TIFFGetA(raster[i*w + j]) << 24 | ((uint32) sumB << 16) | ((uint32) sumG << 8) | ((uint32) sumR);
        }
    }
    delete [] copy; 
}

__global__ void filter_image_cuda(uint32 *raster, uint32 *copy, uint32 w, uint32 h, const float *filter, int f_len, uint32 d, uint32 st, uint32 end_w, uint32 end_h) {    
    
    
	
    
    // applies filter
	
	// Start  Indices
	uint32 start_i = (blockIdx.y * blockDim.y) + threadIdx.y + st;
	uint32 start_j = (blockIdx.x * blockDim.x) + threadIdx.x + st;
 	
    uint32 idx, pixel;
    float sumR, sumG, sumB;
    // applies filter
    
    for (uint32 i = start_i ; i < end_h ; i++) {
        for (uint32 j = start_j ; j < end_w ; j++) {
            sumR = sumG = sumB = 0;
            for (uint32 k = 0 ; k < d ; k ++) {
                idx = (i-st+k)*w + (j-st);
                for (uint32 l = 0 ; l < d ; l++) {
                    pixel = copy[idx++];
                    sumR += (filter[k*d + l] * TIFFGetR(pixel));
                    sumG += (filter[k*d + l] * TIFFGetG(pixel));
                    sumB += (filter[k*d + l] * TIFFGetB(pixel));
                }
            }
            cuda_clamp(&sumR);
            cuda_clamp(&sumG);
            cuda_clamp(&sumB);
            raster[i*w + j] = TIFFGetA(raster[i*w + j]) << 24 | ((uint32) sumB << 16) | ((uint32) sumG << 8) | ((uint32) sumR);
        }
    }
}

void filter_image_par(uint32 *raster, uint32 w, uint32 h, const float *filter, int f_len, int n_threads, int n_blocks) {
    //
    // TODO: here you will filter the image in raster using GPU threads
    //
	
    // Consistent Computations
	uint32 d = (uint32) std::sqrt(f_len);
	uint32 st = d / 2;
    uint32 end_w = w - d/2;
    uint32 end_h = h - d/2;
	uint32 n = w*h;
	
	
	// Create Blocks and threads
	dim3 threadsPerBlock(n_threads, n_threads, 1);
	dim3 numBlocks(n_blocks,n_blocks,1);

	// create pointers for the CUDA arrays
    uint32 *copy_in;
	uint32 *raster_out;
	float *filter_in;
	
	// variable to check for CUDA errors
    hipError_t status;
	
	// choose GPU to run
    status = hipSetDevice(0);
    if (status != hipSuccess) std::cerr << "hipSetDevice failed!" << std::endl;
	
	// allocate space for the arrays in the GPU
    status = hipMalloc(&copy_in, sizeof(uint32) * n);
    if (status != hipSuccess) std::cerr << "hipMalloc (copy_in) failed!" << std::endl;
	status = hipMalloc(&raster_out, sizeof(uint32) * n);
    if (status != hipSuccess) std::cerr << "hipMalloc (raster_out) failed!" << std::endl;
    status = hipMalloc(&filter_in, sizeof(float) * f_len);
    if (status != hipSuccess) std::cerr << "hipMalloc (filter) failed!" << std::endl;
	
	// transfer data from CPU to GPU
    status = hipMemcpy(copy_in, raster, sizeof(uint32) * n, hipMemcpyHostToDevice);
    if (status != hipSuccess) std::cerr << "hipMemcpy H2D failed! - copy" << std::endl;
	status = hipMemcpy(raster_out, raster, sizeof(uint32) * n, hipMemcpyHostToDevice);
    if (status != hipSuccess) std::cerr << "hipMemcpy H2D failed! - raster" << std::endl;
    status = hipMemcpy(filter_in, filter, sizeof(float) * f_len, hipMemcpyHostToDevice);
    if (status != hipSuccess) std::cerr << "hipMemcpy H2D failed! - filter" << std::endl;
	
	// Do the work in the GPU
	//std::cout << "Blocks: " << std::ceil((float)n/THREADS_PER_BLOCK) << std::endl;
	filter_image_cuda<<<numBlocks,threadsPerBlock>>>(raster_out, copy_in, w, h, filter_in, f_len, d, st, end_w, end_h);
	
	// wait for the kernel to finish, and check for errors
    status = hipDeviceSynchronize();
    if (status != hipSuccess) std::cerr << "error code " << status << " returned after kernel!" << std::endl;

	
    // transfer results from GPU to CPU
    status = hipMemcpy(raster, raster_out, sizeof(uint32) * n, hipMemcpyDeviceToHost);
    if (status != hipSuccess) std::cerr << "hipMemcpy D2H failed! - final" << std::endl;
	
	
	// Free memory
	hipFree(copy_in);
	hipFree(raster_out);
	hipFree(filter_in);
}

float *load_filter(const char *fname, int *n) {
    std::ifstream myfile(fname);
    if (! myfile) {
        throw std::runtime_error("Could not open filter file");
    }
    myfile >> *n;
    float *filter = new float[*n];
    for (int i = 0 ; i < *n ; i++) myfile >> filter[i];
    myfile.close();
    return filter;
}

int main(int argc, char* argv[]) {
    if (argc != 7) {
        std::cout << "Usage:\t./filter <in_fname> <out_fname> <filter_fname> <algo>" << std::endl;
        std::cout << "<in_fname> path to the input image" << std::endl;
        std::cout << "<out_fname> path to the output image" << std::endl;
        std::cout << "<filter_fname> path to the filter file" << std::endl;
        std::cout << "<algo> whether to use the sequential (seq) or parallel algorithm (par)" << std::endl;
        std::cout << "<n_threads> number of threads to use (Ex: enter 5 for 25 threads/block)" << std::endl;
        std::cout << "<n_blocks> number of blocks to use [Ex: enter 2 for 4 blocks]" << std::endl;
        return 0;
    }
	
    uint32 width, height;
    
    int n_threads = std::stoi(argv[5]);
    int n_blocks = std::stoi(argv[6]);

    // loads the filter
    int f_len;
    float *filter = load_filter(argv[3], &f_len);
    
    // loads image bytes from file name supplied as a command line argument
    // this function allocates memory dynamically
    uint32 *image = load_tiff(argv[1], &width, &height);

    // measure time of the algorithm
    auto start = std::chrono::high_resolution_clock::now();
    if (! std::strcmp(argv[4], "seq")) {
        // call the sequential implementation
        filter_image_seq(image, width, height, filter, f_len);
    } else if (! std::strcmp(argv[4], "par")) {
        // TODO: call the parallel implementation
		filter_image_par(image, width, height, filter, f_len, n_threads, n_blocks);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << diff.count() << std::endl;   

    // save new file with filtered image
    save_tiff(argv[2], image, width, height);

    // frees memory allocated by load_filter and load_tiff
    delete [] filter;
    _TIFFfree(image);

    return 0;
}